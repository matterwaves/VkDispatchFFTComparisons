#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <chrono>

#define CHECK_CUDA(call) \
    do { \
        hipError_t status = (call); \
        if (status != hipSuccess) { \
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(status)); \
            return 1; \
        } \
    } while (0)

#define CHECK_CUFFT(call) \
    do { \
        hipfftResult status = (call); \
        if (status != HIPFFT_SUCCESS) { \
            fprintf(stderr, "cuFFT error: %d\n", status); \
            return 1; \
        } \
    } while (0)

int main() {
    const int n0 = 16384;      // First dimension
    const int n1 = 256;   // FFT length (middle axis)
    const int n2 = 64;   // Last dimension
    const int n_ffts = 1000;
    const int n_warmup = 100;

    // Total elements
    size_t n_elements = size_t(n0) * n1 * n2;
    size_t buf_size = n_elements * sizeof(hipfftComplex);

    printf("Allocating %.2f GB device buffer...\n", buf_size / 1e9);
    hipfftComplex *d_data;
    CHECK_CUDA(hipMalloc(&d_data, buf_size));
    CHECK_CUDA(hipMemset(d_data, 0, buf_size));
    int rank = 1;  
    int n[] = { n1 };           
    int istride = n2;            
    int idist = 1;             
    int batch = n0 * n2;      

    // This array was defined but not used correctly
    // int inembed[] = { n1, n2 };

    hipfftHandle plan;
    CHECK_CUFFT(hipfftPlanMany(
        &plan,
        rank, 
        n,
        NULL, istride, idist,  // Fixed formatting - removed extra NULL
        NULL, istride, idist,  // Fixed formatting - removed extra NULL
        HIPFFT_C2C,
        batch
    ));

    // Warm-up
    for (int i = 0; i < n_warmup; ++i)
        CHECK_CUFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD));
    CHECK_CUDA(hipDeviceSynchronize());

    // Timing
    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < n_ffts; ++i)
        CHECK_CUFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD));
    CHECK_CUDA(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();

    double elapsed = std::chrono::duration<double>(end - start).count();
    double buffers_per_sec = n_ffts / elapsed;

    printf("Processed %d buffers of size (%d, %d, %d) in %.4f seconds\n",
           n_ffts, n0, n1, n2, elapsed);
    printf("Throughput: %.2f buffers/sec of size (%d, %d, %d)\n",
           buffers_per_sec, n0, n1, n2);

    hipfftDestroy(plan);
    hipFree(d_data);
    return 0;
}
